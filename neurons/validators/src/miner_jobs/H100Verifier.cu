#include "hip/hip_runtime.h"
#include <iostream>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <hiprand/hiprand_kernel.h>
#include <vector>
#include <cstdlib>
#include <ctime>
#include <chrono>
#include <cmath>
#include <fstream>
#include <iomanip>
#include <getopt.h> // For command-line argument parsing
#include <sys/stat.h>  // For mkdir
#include <unistd.h>  // For gethostname() on Linux
#ifdef _WIN32
#include <windows.h> // For GetComputerName() on Windows
#endif

const size_t MEMORY_TEST_SIZE = 1000 * 1024 * 1024;
const uint32_t VALIDATION_VALUE = 0x76543210;

__device__ double lcgRandDevice(unsigned long long seed) {
    // Sophisticated Linear Congruential Generator (LCG) parameters
    const unsigned long long A = 6364136223846793005ULL;  // Multiplier (64-bit)
    const unsigned long long C = 1442695040888963407ULL;  // Increment (64-bit)
    const unsigned long long M = 9223372036854775807ULL;  // Modulus (2^63 - 1)

    // Update the seed using the LCG formula
    seed = (A * seed + C) % M;

    // Apply a floating-point transformation to scale to [0, 1)
    double rand_val = static_cast<double>(seed) / static_cast<double>(M);

    // Return a value in the range [0, 1)
    return rand_val;
}

__global__ void MatrixMultiply(double *A, double *B, double *C, long n, long k) {
    long row = blockIdx.y * blockDim.y + threadIdx.y;
    long col = blockIdx.x * blockDim.x + threadIdx.x;

    // Ensure within matrix bounds
    if (row < n && col < n) {
        double value = 0.0;
        // Perform the matrix multiplication (dot product of row of A and column of B)
        for (int i = 0; i < k; i++) {
            value += A[row * k + i] * B[i * n + col];
        }

        double index_factor = (row * n + col + value) / (double)(n * n);  // Normalize between 0 and 1
        double index_increase = index_factor * n * 1.5;  // Higher increase for smaller index

        // Store the result in C
        C[row * n + col] = value + index_increase;
    }
}

__global__ void GenerateRandomMatrix(double* A, long n, long k, unsigned long long seed) {
    // int idx = blockIdx.x * blockDim.x + threadIdx.x;
    // int idy = blockIdx.y * blockDim.y + threadIdx.y;
    long idx = blockIdx.y * blockDim.y + threadIdx.x;
    long idy = blockIdx.x * blockDim.x + threadIdx.y;

    // printf("blockDim.x: %d, blockDim.y: %d, blockIdx.x: %d , blockIdx.y: %d, threadIdx.x: %d, threadIdx.y: %d, Row: %d, Col: %d \n", 
        // blockDim.x, blockDim.y,  blockIdx.x, blockIdx.y, threadIdx.x ,threadIdx.y, idx, idy);
    long divider = 1;

    if (n > 100 || k > 100) {
        divider = 10;
    }

    if (idx < n && idy < k) {
        // Generate more distinct random numbers by using different offsets for each thread
        unsigned long long seed_a = seed + idy * n + idx;  // Unique seed for each thread in matrix A

        double rand_num_a = lcgRandDevice(seed_a) / divider;

        if (idx < n && idy < k) {
            // printf("index: %d , Value: %f, Row: %d, Col: %d \n", idy * n + idx, lcgRandDevice(seed_a), idx, idy);
            A[idy * n + idx] = rand_num_a;  // A is of size n * k
        }
    }
}

std::string getComputerName() {
    char buffer[256];
#ifdef _WIN32
    // Windows-specific code
    DWORD size = sizeof(buffer);
    if (GetComputerNameA(buffer, &size)) {
        return std::string(buffer);
    } else {
        return "Unknown";
    }
#else
    // Linux-specific code
    if (gethostname(buffer, sizeof(buffer)) == 0) {
        return std::string(buffer);
    } else {
        return "Unknown";
    }
#endif
}

std::string getCurrentDateTime() {
    // Get current time
    std::time_t t = std::time(nullptr);  
    std::tm tm = *std::localtime(&t);  // Convert to local time
    
    // Format the date and time as YYYY_MM_DD HH_MM
    std::ostringstream oss;
    oss << std::put_time(&tm, "%Y_%m_%d_%H_%M");
    
    return oss.str();
}

void writeToResultFile(long N, long K, long seed, float bandwidth, double *mulMatrix, const std::string& result_path) {
    std::ofstream outFile(result_path);

    if (!outFile) {
        std::cerr << "Error opening file for writing!" << std::endl;
        return;
    }

    outFile << "Dimension N: " << N << ", K: " << K << std::endl;

    outFile << "Matrix:\n";
    for (long i = 0; i < N; ++i) {
        for (long j = 0; j < N; ++j) {
            outFile << std::fixed << std::setprecision(2) << mulMatrix[i * N + j] << " ";  // 2 decimal precision
        }
        outFile << "\n";
    }

    // Write the bandwidth to the file
    outFile << "Bandwidth: " << bandwidth << std::endl;

    // Close the file
    outFile.close();
    std::cout << "Validation Results saved to: " << result_path << std::endl;
}

bool isGPUAvailable(int deviceId) {
    hipError_t err = hipSetDevice(deviceId);
    if (err != hipSuccess) {
        std::cerr << "GPU " << deviceId << " not available: " << hipGetErrorString(err) << std::endl;
        return false;
    }

    err = hipDeviceSynchronize();
    if (err != hipSuccess) {
        std::cerr << "Error synchronizing GPU " << deviceId << ": " << hipGetErrorString(err) << std::endl;
        return false;
    }

    return true;
}

int findAvailableGPU() {
    int deviceCount = 0;
    hipError_t err = hipGetDeviceCount(&deviceCount);
    if (err != hipSuccess) {
        std::cerr << "CUDA error: " << hipGetErrorString(err) << std::endl;
        return -1; // Error, return invalid GPU id
    }

    // Iterate over all available GPUs and find one that is available
    for (int deviceId = 0; deviceId < deviceCount; ++deviceId) {
        if (isGPUAvailable(deviceId)) {
            // std::cout << "Using GPU: " << deviceId << std::endl;
            return deviceId;  // Return first available GPU
        }
    }

    std::cerr << "No available GPUs found!" << std::endl;
    return -1;
}

class H100Verifier {
public:
    H100Verifier(long m_dim_n, long m_dim_k) {
        this->m_dim_n = m_dim_n;
        this->m_dim_k = m_dim_k;
        m_MulMatrix = new double[m_dim_n * m_dim_n];
    }

    ~H100Verifier() {
        delete[] m_MulMatrix;
    }

    void testBandWidth() {
        // Performance test
        int deviceId = findAvailableGPU();
        if (deviceId == -1) return;

        hipSetDevice(deviceId); // Set the chosen GPU
        

        hipDeviceptr_t d_perfTest;  // Declare d_perfTest here
        hipMalloc(&d_perfTest, MEMORY_TEST_SIZE);
        
        // Create events for timing
        hipEvent_t start, stop;  // Declare timing events here
        float elapsedTime = 0;
        hipEventCreateWithFlags(&start, hipEventDefault);
        hipEventCreateWithFlags(&stop, hipEventDefault);

        // Time memory operations
        hipEventRecord(start, 0);
        hipMemsetD32(d_perfTest, VALIDATION_VALUE, MEMORY_TEST_SIZE / 4);
        hipEventRecord(stop, 0);
        hipEventSynchronize(stop);
        hipEventElapsedTime(&elapsedTime, start, stop);

        // Calculate and verify bandwidth
        m_bandWidth = (MEMORY_TEST_SIZE / (elapsedTime * 0.001)) * (1.0f / 1e9f);
    }

    void printMatrixFromDevice(double* d_A, long n, long k) {
        // Allocate memory on the host to hold the matrix
        double* h_A = new double[n * k];

        // Copy the matrix from device to host
        hipError_t err = hipMemcpy(h_A, d_A, n * k * sizeof(double), hipMemcpyDeviceToHost);
        if (err != hipSuccess) {
            std::cerr << "printMatrixFromDevice Error: " << hipGetErrorString(err) << std::endl;
            delete[] h_A;
            return;
        }

        // Print the matrix
        for (long i = 0; i < n; i++) {
            for (long j = 0; j < k; j++) {
                std::cout << std::fixed << std::setprecision(2) << h_A[i * k + j] << " ";  // Print with 2 decimal points
            }
            std::cout << std::endl;
        }

        // Free the host memory
        delete[] h_A;
    }

    void generateMatrix(double* matrix, long n, long k, unsigned long long seed) {
        dim3 threadsPerBlock(32, 32); // 1024 threads per block (32x32 block of threads)

        long temp = n;
        if (n > k) {
            n = k;
            k = temp;
        }

        std::cout << "n:" << n<< std::endl;
        std::cout << "k:" << k << std::endl;

        dim3 numBlocks((k + threadsPerBlock.x - 1) / threadsPerBlock.x, 
                       (n + threadsPerBlock.y - 1) / threadsPerBlock.y);
        
        // numBlocks.x = min(numBlocks.x, 65535); // Ensure numBlocks.x does not exceed the max allowed
        numBlocks.y = min(numBlocks.y, 65535); // Ensure numBlocks.y does not exceed the max allowed
        
        std::cout << "Num Block x:" <<  numBlocks.x << std::endl;
        std::cout << "Num Block y:" <<  numBlocks.y << std::endl;
        
        GenerateRandomMatrix<<<numBlocks, threadsPerBlock>>>(matrix, n, k, seed);
        hipDeviceSynchronize();
        
        hipError_t err = hipGetLastError();
        if (err != hipSuccess) {
            std::cerr << "generateMatrix CUDA Error in kernel launch: " << hipGetErrorString(err) << std::endl;
        }
    }
    
    int verifyChallenge(unsigned long long seed = 1234, const std::string& result_path = "validate_result.txt") {
        double *d_A, *d_B, *d_C;
        hipError_t err;
        getMaxMatrixDimensions();
        int deviceId = findAvailableGPU();
        if (deviceId == -1) return -1;
        // deviceId = 1;
        hipSetDevice(deviceId); // Set the chosen GPU

        size_t free_mem, total_mem;
        hipMemGetInfo(&free_mem, &total_mem);
        std::cout << "Free memory: " << free_mem / (1024 * 1024 * 1024) << " GB" << std::endl;
        std::cout << "Total memory: " << total_mem / (1024 * 1024 * 1024) << " GB" << std::endl;

        // Allocate memory on device
        std::cout << "Generating Matrix A" << std::endl;
        err = hipMalloc(&d_A, m_dim_n * m_dim_k * sizeof(double));
        if (err != hipSuccess) return handleCudaError(err);
        std::cout << "Generating Matrix B" << std::endl;
        err = hipMalloc(&d_B, m_dim_k * m_dim_n * sizeof(double));
        if (err != hipSuccess) return handleCudaError(err);
        std::cout << "Generating Matrix C" << std::endl;
        err = hipMalloc(&d_C, m_dim_n * m_dim_n * sizeof(double));
        if (err != hipSuccess) return handleCudaError(err);
    
        generateMatrix(d_A, m_dim_n, m_dim_k, seed);
        generateMatrix(d_B, m_dim_k, m_dim_n, seed + m_dim_k * m_dim_n);

        #if DEBUG
            // std::cout << "Matrix A:" << std::endl;
            // printMatrixFromDevice(d_A, m_dim_n, m_dim_k);
            // std::cout << "Matrix B:" << std::endl;
            // printMatrixFromDevice(d_B, m_dim_k, m_dim_n);
        #endif
    
        testBandWidth();

        // Check for any errors during kernel launch
        err = hipDeviceSynchronize();
        if (err != hipSuccess) return handleCudaError(err);
        
        // Launch kernel for matrix multiplication C = A * B
        dim3 threadsPerBlock(32, 32);
        dim3 numBlocks((m_dim_n + threadsPerBlock.x - 1) / threadsPerBlock.x, (m_dim_n + threadsPerBlock.y - 1) / threadsPerBlock.y);
        std::cout << "Matrix multiply:" << std::endl;
        MatrixMultiply<<<numBlocks, threadsPerBlock>>>(d_A, d_B, d_C, m_dim_n, m_dim_k);
        // Check for errors during kernel launch

        hipDeviceSynchronize();
        err = hipGetLastError();
        if (err != hipSuccess) return handleCudaError(err);
    
        // Copy result matrix C back to the host
        err = hipMemcpy(m_MulMatrix, d_C, m_dim_n * m_dim_n * sizeof(double), hipMemcpyDeviceToHost);
        if (err != hipSuccess) return handleCudaError(err);

        // Free device memory
        hipFree(d_A);
        hipFree(d_B);
        hipFree(d_C);
        
        writeToResultFile(m_dim_n, m_dim_k, seed, m_bandWidth, m_MulMatrix, result_path);
    
        return 0;
    }

    void getMaxMatrixDimensions() {
        int deviceId = 0; // Default to first GPU
        hipDeviceProp_t props;
    
        // Get device properties
        hipError_t err = hipGetDeviceProperties(&props, deviceId);
        if (err != hipSuccess) {
            std::cerr << "Failed to get device properties: " << hipGetErrorString(err) << std::endl;
            return;
        }
    
        // Get available memory
        size_t freeMemory, totalMemory;

        err = hipMemGetInfo(&freeMemory, &totalMemory);
        if (err != hipSuccess) {
            std::cerr << "Failed to get memory info: " << hipGetErrorString(err) << std::endl;
            return;
        }
    
        std::cout << "Free GPU Memory: " << freeMemory / (1024.0 * 1024.0) << " MB" << std::endl;
        std::cout << "Total GPU Memory: " << totalMemory / (1024.0 * 1024.0) << " MB" << std::endl;
        
        const long maxMemory = (long)totalMemory - 1048576000 * 2;  // Reduce total memory by 1 GB to leave some space for overhead
        std::cout << "Available GPU Memory: " << maxMemory / (1024.0 * 1024.0) << " MB" << std::endl;

        // Assuming double precision (8 bytes per element)
        const long elementSize = sizeof(double);
        
        // Calculate maximum matrix dimensions n * m that fit in free memory
        long maxElements = maxMemory / elementSize;  // Max number of elements we can allocate
        
        // Let's assume square matrix for simplicity (n = m)
        long max_m_dim_k = maxElements / (2 * m_dim_n) - m_dim_n;
        // m_dim_k = max_m_dim_k;

        std::cout << "Max matrix dimension (n = m): " << m_dim_n << " x " << max_m_dim_k << std::endl;
    
        // If you want to handle non-square matrices, you can change the logic accordingly.
        // For example, you could use a different approach to split the available memory into n and m.
    }

    double* getMulMatrix() const {
        return m_MulMatrix;
    }

    float getBandWidth() {
        return m_bandWidth;
    }

private:
    long m_dim_n;
    long m_dim_k;
    double* m_MulMatrix;
    unsigned long m_seed;
    float m_bandWidth = 0.0f;

    int handleCudaError(hipError_t err) const {
        std::cerr << "CUDA Error: " << hipGetErrorString(err) << std::endl;
        return -1;
    }
};


class ArgumentParser {
public:
    static void parseArguments(int argc, char* argv[], long& dim_n, long& dim_k, unsigned long long& seed, std::string& result_path) {
        static struct option long_options[] = {
            {"dim_n", required_argument, 0, 'n'},
            {"dim_k", required_argument, 0, 'k'},
            {"seed", required_argument, 0, 's'},
            {"result_path", required_argument, 0, 'r'}, // New argument for result path
            {0, 0, 0, 0}
        };

        int option_index = 0;
        int opt;
        while ((opt = getopt_long(argc, argv, "", long_options, &option_index)) != -1) {
            switch (opt) {
                case 'n':
                    dim_n = std::atoi(optarg);
                    break;
                case 'k':
                    dim_k = std::atoi(optarg);
                    break;
                case 's':
                    seed = std::strtoull(optarg, nullptr, 10);
                    break;
                case 'r':
                    result_path = std::string(optarg);  // Store result path
                    break;
                default:
                    std::cerr << "Usage: --dim_n <value> --dim_k <value> --seed <value> --result_path <path>" << std::endl;
                    exit(EXIT_FAILURE);
            }
        }
    }
};
    
int main(int argc, char* argv[]) {
    long N = 1000, K = 5176864;
    unsigned long long seed = 234;
    std::string result_path = "validate_result.txt";

    ArgumentParser::parseArguments(argc, argv, N, K, seed, result_path);

    std::cout << "Seed: " << seed << std::endl;
    
    H100Verifier verifier(N, K);
    auto start = std::chrono::high_resolution_clock::now();
    int result = verifier.verifyChallenge(seed, result_path);
    auto end = std::chrono::high_resolution_clock::now();
    std::chrono::duration<double> duration = end - start;
    std::cout << "Execution time on GPU: " << duration.count() << " seconds." << std::endl;

    return result;
}