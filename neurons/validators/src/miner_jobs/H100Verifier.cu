#include <iostream>
#include <hip/hip_runtime.h>

#include <hiprand/hiprand_kernel.h>
#include <vector>
#include <cstdlib>
#include <ctime>
#include <chrono>
#include <cmath>
#include <fstream>
#include <iomanip>
#include <getopt.h> // For command-line argument parsing
#include <sys/stat.h>  // For mkdir
#include <unistd.h>  // For gethostname() on Linux
#ifdef _WIN32
#include <windows.h> // For GetComputerName() on Windows
#endif

const size_t MEMORY_TEST_SIZE = 1000 * 1024 * 1024;
const uint32_t VALIDATION_VALUE = 0x76543210;

__device__ double lcgRandDevice(unsigned long long seed) {
    // Sophisticated Linear Congruential Generator (LCG) parameters
    const unsigned long long A = 6364136223846793005ULL;  // Multiplier (64-bit)
    const unsigned long long C = 1442695040888963407ULL;  // Increment (64-bit)
    const unsigned long long M = 9223372036854775807ULL;  // Modulus (2^63 - 1)

    // Update the seed using the LCG formula
    seed = (A * seed + C) % M;

    // Apply a floating-point transformation to scale to [0, 1)
    double rand_val = static_cast<double>(seed) / static_cast<double>(M);

    // Return a value in the range [0, 1)
    return rand_val;
}

__global__ void MatrixMultiply(double *A, double *B, double *C, int n, int k) {
    int row = blockIdx.y * blockDim.y + threadIdx.y;
    int col = blockIdx.x * blockDim.x + threadIdx.x;

    // Ensure within matrix bounds
    if (row < n && col < n) {
        double value = 0.0;
        // Perform the matrix multiplication (dot product of row of A and column of B)
        for (int i = 0; i < k; i++) {
            value += A[row * k + i] * B[i * n + col];
        }

        double index_factor = (row * n + col + value) / (double)(n * n);  // Normalize between 0 and 1
        double index_increase = index_factor * n * 1.5;  // Higher increase for smaller index

        // Store the result in C
        C[row * n + col] = value + index_increase;
    }
}

__global__ void GenerateRandomMatrix(double* A, int n, int k, unsigned long long seed) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    int idy = blockIdx.y * blockDim.y + threadIdx.y;

    int divider = 1;

    if (n > 100 || k > 100) {
        divider = 10;
    }

    if (idx < n && idy < k) {
        // Generate more distinct random numbers by using different offsets for each thread
        unsigned long long seed_a = seed + idy * n + idx;  // Unique seed for each thread in matrix A

        double rand_num_a = lcgRandDevice(seed_a) / divider;

        if (idx < n && idy < k) {
            // printf("index: %d , Value: %f, Row: %d, Col: %d \n", idy * n + idx, lcgRandDevice(seed_a), idx, idy);
            A[idy * n + idx] = rand_num_a;  // A is of size n * k
        }
    }
}

std::string getComputerName() {
    char buffer[256];
#ifdef _WIN32
    // Windows-specific code
    DWORD size = sizeof(buffer);
    if (GetComputerNameA(buffer, &size)) {
        return std::string(buffer);
    } else {
        return "Unknown";
    }
#else
    // Linux-specific code
    if (gethostname(buffer, sizeof(buffer)) == 0) {
        return std::string(buffer);
    } else {
        return "Unknown";
    }
#endif
}

std::string getCurrentDateTime() {
    // Get current time
    std::time_t t = std::time(nullptr);  
    std::tm tm = *std::localtime(&t);  // Convert to local time
    
    // Format the date and time as YYYY_MM_DD HH_MM
    std::ostringstream oss;
    oss << std::put_time(&tm, "%Y_%m_%d_%H_%M");
    
    return oss.str();
}

void writeToResultFile(int N, int K, long seed, float bandwidth, double *mulMatrix, const std::string& result_path) {
    std::ofstream outFile(result_path);

    if (!outFile) {
        std::cerr << "Error opening file for writing!" << std::endl;
        return;
    }

    outFile << "Dimension N: " << N << ", K: " << K << std::endl;

    outFile << "Matrix:\n";
    for (int i = 0; i < N; ++i) {
        for (int j = 0; j < N; ++j) {
            outFile << std::fixed << std::setprecision(2) << mulMatrix[i * N + j] << " ";  // 2 decimal precision
        }
        outFile << "\n";
    }

    // Write the bandwidth to the file
    outFile << "Bandwidth: " << bandwidth << std::endl;

    // Close the file
    outFile.close();
    std::cout << "Validation Results saved to: " << result_path << std::endl;
}

bool isGPUAvailable(int deviceId) {
    hipError_t err = hipSetDevice(deviceId);
    if (err != hipSuccess) {
        std::cerr << "GPU " << deviceId << " not available: " << hipGetErrorString(err) << std::endl;
        return false;
    }

    err = hipDeviceSynchronize();
    if (err != hipSuccess) {
        std::cerr << "Error synchronizing GPU " << deviceId << ": " << hipGetErrorString(err) << std::endl;
        return false;
    }

    return true;
}

int findAvailableGPU() {
    int deviceCount = 0;
    hipError_t err = hipGetDeviceCount(&deviceCount);
    if (err != hipSuccess) {
        std::cerr << "CUDA error: " << hipGetErrorString(err) << std::endl;
        return -1; // Error, return invalid GPU id
    }

    // Iterate over all available GPUs and find one that is available
    for (int deviceId = 0; deviceId < deviceCount; ++deviceId) {
        if (isGPUAvailable(deviceId)) {
            // std::cout << "Using GPU: " << deviceId << std::endl;
            return deviceId;  // Return first available GPU
        }
    }

    std::cerr << "No available GPUs found!" << std::endl;
    return -1;
}

class H100Verifier {
public:
    H100Verifier(int m_dim_n, int m_dim_k) {
        this->m_dim_n = m_dim_n;
        this->m_dim_k = m_dim_k;
        m_MulMatrix = new double[m_dim_n * m_dim_n];
    }

    ~H100Verifier() {
        delete[] m_MulMatrix;
    }

    void testBandWidth() {
        // Performance test
        int deviceId = findAvailableGPU();
        if (deviceId == -1) return;

        hipSetDevice(deviceId); // Set the chosen GPU
        

        hipDeviceptr_t d_perfTest;  // Declare d_perfTest here
        hipMalloc(&d_perfTest, MEMORY_TEST_SIZE);
        
        // Create events for timing
        hipEvent_t start, stop;  // Declare timing events here
        float elapsedTime = 0;
        hipEventCreateWithFlags(&start, hipEventDefault);
        hipEventCreateWithFlags(&stop, hipEventDefault);

        // Time memory operations
        hipEventRecord(start, 0);
        hipMemsetD32(d_perfTest, VALIDATION_VALUE, MEMORY_TEST_SIZE / 4);
        hipEventRecord(stop, 0);
        hipEventSynchronize(stop);
        hipEventElapsedTime(&elapsedTime, start, stop);

        // Calculate and verify bandwidth
        m_bandWidth = (MEMORY_TEST_SIZE / (elapsedTime * 0.001)) * (1.0f / 1e9f);
    }

    void printMatrixFromDevice(double* d_A, int n, int k) {
        // Allocate memory on the host to hold the matrix
        double* h_A = new double[n * k];

        // Copy the matrix from device to host
        hipError_t err = hipMemcpy(h_A, d_A, n * k * sizeof(double), hipMemcpyDeviceToHost);
        if (err != hipSuccess) {
            std::cerr << "CUDA Error: " << hipGetErrorString(err) << std::endl;
            delete[] h_A;
            return;
        }

        // Print the matrix
        for (int i = 0; i < n; i++) {
            for (int j = 0; j < k; j++) {
                std::cout << std::fixed << std::setprecision(2) << h_A[i * k + j] << " ";  // Print with 2 decimal points
            }
            std::cout << std::endl;
        }

        // Free the host memory
        delete[] h_A;
    }

    void generateMatrix(double* matrix, int n, int k, unsigned long long seed) {
        dim3 threadsPerBlock(16, 16); // 16x16 block of threads
        dim3 numBlocks((n + threadsPerBlock.x - 1) / threadsPerBlock.x, (k + threadsPerBlock.y - 1) / threadsPerBlock.y);

        GenerateRandomMatrix<<<numBlocks, threadsPerBlock>>>(matrix, n, k, seed);
        hipDeviceSynchronize();
    }
    
    int verifyChallenge(unsigned long long seed = 1234, const std::string& result_path = "validate_result.txt") {
        double *d_A, *d_B, *d_C;
        hipError_t err;
        getMaxMatrixDimensions();
        int deviceId = findAvailableGPU();
        if (deviceId == -1) return -1;
        // deviceId = 1;
        hipSetDevice(deviceId); // Set the chosen GPU

        // Allocate memory on device
        err = hipMalloc(&d_A, m_dim_n * m_dim_k * sizeof(double));
        if (err != hipSuccess) return handleCudaError(err);
    
        err = hipMalloc(&d_B, m_dim_k * m_dim_n * sizeof(double));
        if (err != hipSuccess) return handleCudaError(err);
    
        err = hipMalloc(&d_C, m_dim_n * m_dim_n * sizeof(double));
        if (err != hipSuccess) return handleCudaError(err);
    
        generateMatrix(d_A, m_dim_n, m_dim_k, seed);
        generateMatrix(d_B, m_dim_k, m_dim_n, seed + m_dim_k * m_dim_n);

        #if DEBUG
            // std::cout << "Matrix A:" << std::endl;
            // printMatrixFromDevice(d_A, m_dim_n, m_dim_k);
            // std::cout << "Matrix B:" << std::endl;
            // printMatrixFromDevice(d_B, m_dim_k, m_dim_n);
        #endif
    
        testBandWidth();

        // Check for any errors during kernel launch
        err = hipDeviceSynchronize();
        if (err != hipSuccess) return handleCudaError(err);
        
        // Launch kernel for matrix multiplication C = A * B
        dim3 threadsPerBlock(16, 16);
        dim3 numBlocks((m_dim_n + threadsPerBlock.x - 1) / threadsPerBlock.x, (m_dim_n + threadsPerBlock.y - 1) / threadsPerBlock.y);

        MatrixMultiply<<<numBlocks, threadsPerBlock>>>(d_A, d_B, d_C, m_dim_n, m_dim_k);
        // Check for errors during kernel launch

        hipDeviceSynchronize();
        // err = cudaGetLastError();
        // if (err != cudaSuccess) return handleCudaError(err);
    
        // Copy result matrix C back to the host
        err = hipMemcpy(m_MulMatrix, d_C, m_dim_n * m_dim_n * sizeof(double), hipMemcpyDeviceToHost);
        if (err != hipSuccess) return handleCudaError(err);

        // Free device memory
        hipFree(d_A);
        hipFree(d_B);
        hipFree(d_C);
        
        writeToResultFile(m_dim_n, m_dim_k, seed, m_bandWidth, m_MulMatrix, result_path);
    
        return 0;
    }

    void getMaxMatrixDimensions() {
        int deviceId = 0; // Default to first GPU
        hipDeviceProp_t props;
    
        // Get device properties
        hipError_t err = hipGetDeviceProperties(&props, deviceId);
        if (err != hipSuccess) {
            std::cerr << "Failed to get device properties: " << hipGetErrorString(err) << std::endl;
            return;
        }
    
        // Get available memory
        size_t freeMemory, totalMemory;

        err = hipMemGetInfo(&freeMemory, &totalMemory);
        if (err != hipSuccess) {
            std::cerr << "Failed to get memory info: " << hipGetErrorString(err) << std::endl;
            return;
        }
    
        std::cout << "Free GPU Memory: " << freeMemory / (1024.0 * 1024.0) << " MB" << std::endl;
        std::cout << "Total GPU Memory: " << totalMemory / (1024.0 * 1024.0) << " MB" << std::endl;
        
        const long maxMemory = (long)totalMemory - 1048576000 * 2;  // Reduce total memory by 1 GB to leave some space for overhead
        std::cout << "Available GPU Memory: " << maxMemory / (1024.0 * 1024.0) << " MB" << std::endl;

        // Assuming double precision (8 bytes per element)
        const long elementSize = sizeof(double);
        
        // Calculate maximum matrix dimensions n * m that fit in free memory
        long maxElements = maxMemory / elementSize;  // Max number of elements we can allocate
        
        // Let's assume square matrix for simplicity (n = m)
        m_dim_k = maxElements / (2 * m_dim_n) - m_dim_n;
        
        std::cout << "Max matrix dimension (n = m): " << m_dim_n << " x " << m_dim_k << std::endl;
    
        // If you want to handle non-square matrices, you can change the logic accordingly.
        // For example, you could use a different approach to split the available memory into n and m.
    }

    double* getMulMatrix() const {
        return m_MulMatrix;
    }

    float getBandWidth() {
        return m_bandWidth;
    }

private:
    int m_dim_n;
    int m_dim_k;
    double* m_MulMatrix;
    unsigned long m_seed;
    float m_bandWidth = 0.0f;

    int handleCudaError(hipError_t err) const {
        std::cerr << "CUDA Error: " << hipGetErrorString(err) << std::endl;
        return -1;
    }
};


class ArgumentParser {
public:
    static void parseArguments(int argc, char* argv[], int& dim_n, int& dim_k, unsigned long long& seed, std::string& result_path) {
        static struct option long_options[] = {
            {"dim_n", required_argument, 0, 'n'},
            {"dim_k", required_argument, 0, 'k'},
            {"seed", required_argument, 0, 's'},
            {"result_path", required_argument, 0, 'r'}, // New argument for result path
            {0, 0, 0, 0}
        };

        int option_index = 0;
        int opt;
        while ((opt = getopt_long(argc, argv, "", long_options, &option_index)) != -1) {
            switch (opt) {
                case 'n':
                    dim_n = std::atoi(optarg);
                    break;
                case 'k':
                    dim_k = std::atoi(optarg);
                    break;
                case 's':
                    seed = std::strtoull(optarg, nullptr, 10);
                    break;
                case 'r':
                    result_path = std::string(optarg);  // Store result path
                    break;
                default:
                    std::cerr << "Usage: --dim_n <value> --dim_k <value> --seed <value> --result_path <path>" << std::endl;
                    exit(EXIT_FAILURE);
            }
        }
    }
};
    
int main(int argc, char* argv[]) {
    int N = 500, K = 3100000;
    unsigned long long seed = 234;
    std::string result_path = "validate_result.txt";

    ArgumentParser::parseArguments(argc, argv, N, K, seed, result_path);

    std::cout << "Seed: " << seed << std::endl;
    
    H100Verifier verifier(N, K);
    auto start = std::chrono::high_resolution_clock::now();
    int result = verifier.verifyChallenge(seed, result_path);
    auto end = std::chrono::high_resolution_clock::now();
    std::chrono::duration<double> duration = end - start;
    std::cout << "Execution time on GPU: " << duration.count() << " seconds." << std::endl;

    return result;
}
